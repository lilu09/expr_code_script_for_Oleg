
#include <iostream>
#include <assert.h>
#include <lib/matrix_mul_tunable.h>
#include <lib/tuplex.h>

using namespace std;

int main(int argc, char* argv[])
{

	assert( hipDeviceReset() == hipSuccess ) ;

	size_t const num_random_points=atoi(argv[1]),
	       num_trys_per_point=atoi(argv[2]),
	       total_measurements=num_trys_per_point*MATRIX_MUL_NUM_VARIANTS*num_random_points;

	//rpid, x1, x2, x3
	vector<tuple<size_t, size_t, size_t, size_t> > random_points;
	random_points.reserve(num_random_points);

	//restore random_points from file
	ifstream file("../1.initialization_db_and_output_points/random_points_production_run.csv");
	file_copy(file, random_points);


	/*std::copy(random_points.begin(), random_points.end(),*/
			/*tuple_ostream_iterator(std::cout, ""));*/
	/*cout<<(char)8<<(char)8<<' '<<endl;*/



	//exprid, seq_id, rpid, vid, value
	vector< tuple<size_t, size_t, size_t, size_t, meterpu::CPU_Time::ResultType> >
		measurements(total_measurements);

	size_t exprid=0;


	using namespace meterpu;
	meter<CPU_Time> cpu_meter;
	meter<CUDA_Time> cuda_meter;


	ofstream outfile("all_measurements.csv");
	tuple_ofstream_iterator start(outfile, "");

	auto i=measurements.begin();
	decltype(i) begin, end;

	// Loop all points
	for(auto p= random_points.begin(); p < random_points.end(); ++p){

		//Can use std::tie here, but I just want references
		const size_t& HA=get<1>(*p), &WA=get<2>(*p), &WB=get<3>(*p);

		vectorpu::vector<float> A(WA*HA,1), B(WA*WB,1), C(HA*WB,0), C_ref(HA*WB,WA);



		cout<<"computing "<<HA<<" "<<WA<<" "<<WB<<"..."<<endl;

		begin=i;



		// For each variant
		for(size_t v=0; v<MATRIX_MUL_NUM_VARIANTS;
				++v){

			// For each repeated try on the same point
			for(size_t r=0; r< num_trys_per_point; ++r){


				if(v<MATRIX_MUL_NUM_VARIANTS-1){
					cpu_meter.start();
					(*matrix_mul_dispatch_table[v])(R(A), R(B), W(C), HA, WA, WB);
					cpu_meter.stop();
					cpu_meter.calc();
					*(i++)
						=make_tuple(exprid++,r,p-random_points.begin(),v,cpu_meter.get_value() );
				}
				else{
					cuda_meter.start();
					(*matrix_mul_dispatch_table[v])(GR(A), GR(B), GW(C), HA, WA, WB);
					cuda_meter.stop();
					cuda_meter.calc();
					*(i++)
						=make_tuple(exprid++,r,p-random_points.begin(),v,cuda_meter.get_value() );
				}



				assert( equal(RI(C),REI(C), RI(C_ref)) );

			}
		}

		end=i;



	}


	copy(measurements.begin(), measurements.end(), start);





	return EXIT_SUCCESS;
}
