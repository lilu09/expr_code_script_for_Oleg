#include "hip/hip_runtime.h"


#include <bits/stdc++.h>
#include <lib/matrix_mul_tunable.h>
#include <lib/tuneit.h>
#include <lib/tuplex.h>

using namespace std;

int main(int argc, char* argv[])
{

	assert(argc==4);

	size_t const num_random_points=atoi(argv[1]),
	       num_trys_per_point=atoi(argv[2]),
	       num_depth=atoi(argv[3]);

	assert( hipDeviceReset() == hipSuccess ) ;

	using namespace std;



	ifstream file("../1.initialization_db_and_output_points/random_points_production_run.csv");
	//id, x1, x2, x3
	vector<tuple<size_t, size_t, size_t, size_t> > random_points;
	random_points.reserve(num_random_points);

	vector<size_t> predictions;
	predictions.reserve(num_random_points);

	//depth, sampled_points_count
	vector<tuple<size_t, size_t> > sampled_points_count;
	sampled_points_count.reserve(num_depth-1);

	file_copy(file, random_points);


	vector<bool> mask(4,true);
	constexpr size_t num_variant=MATRIX_MUL_NUM_VARIANTS;
	constexpr size_t num_vertices=8; //manually calculate pow(2,Dim), ugly!  nvcc can not integrate with gcc on c++14!

	//test from 1 to 11
	for(size_t depth=1; depth<=num_depth;++depth){

		cout<<"Now start depth "<<depth<<"..."<<endl;

		tuneit::tuneit_settings<MATRIX_MUL_NUM_DIM, MATRIX_MUL_NUM_VARIANTS> st{depth, mask, true, false, true, 40, {{1,100}, {1,100}, {1,100}} };

		tuneit::tuneit< num_variant, num_vertices, matrix_mul_tunable<float, size_t, size_t, size_t>,
			float, size_t, size_t, size_t> mytuner(st);

		mytuner.train();

		sampled_points_count.emplace_back( std::make_tuple<size_t,
				size_t>(static_cast<size_t>(depth),
					static_cast<size_t>(mytuner.sampled_points_count()) ) );
		predictions.clear();

		for(auto const &i:random_points){
			predictions.emplace_back( mytuner.predict(get<1>(i), get<2>(i), get<3>(i)) );
		}

		//build a new tuple for output and write to file
		//id, x1, x2, x3, predicted winner
		vector<tuple<size_t, size_t, size_t, size_t, size_t> > final_form;
		final_form.reserve(random_points.size());
		for(size_t i=0; i<random_points.size();++i){
			final_form.emplace_back( std::make_tuple<size_t, size_t, size_t, size_t, size_t> (
						static_cast<size_t>(get<0>(random_points[i])),
						static_cast<size_t>(get<1>(random_points[i])),
						static_cast<size_t>(get<2>(random_points[i])),
						static_cast<size_t>(get<3>(random_points[i])),
						static_cast<size_t>(predictions[i])
						) );
		}

		ofstream file2( ("predictions_depth_"+to_string(depth)+".csv").c_str() );
		tuple_ofstream_iterator start2(file2, "");
		copy(final_form.begin(), final_form.end(), start2  );

		ofstream file3("depth_and_sampled_points_count.csv");
		tuple_ofstream_iterator start3(file3, "");
		copy(sampled_points_count.begin(), sampled_points_count.end(), start3);

	}

	//write <depth, sampled_points_count> to table
	/*vector<tuple<size_t, size_t> > final_form;*/

	/*for(size_t depth=1; depth<11;++depth){*/
	/*final_form.emplace_back(*/
	/*std::make_tuple<size_t, size_t>(depth,*/
	/*sampled_points_count[depth-1])*/
	/*);*/
	/*}*/



	return EXIT_SUCCESS;
}
